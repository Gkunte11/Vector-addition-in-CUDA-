#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<time.h>


__global__ void addition(float *d_a, float *d_b, float *d_c, int n)
{
	
	// kernel function for calculating vector addition. blockIdx.x determines the block number, blockDim.x determines the number of threads per block and
	// threadIdx.x tells us the thread number in a particular block

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < n)
	{
		d_c[i] = d_a[i] + d_b[i];
	}
}

int main()
{
	int n;
	printf("******* GPU Vector Addition *******\n");
	printf("Enter the total numbers: ");
	scanf("%d", &n);

	float *h_a, *h_b, *h_c;
	float *d_a, *d_b, *d_c;

	size_t bytes = n * sizeof(float);

	// dynamically allocating size to the device and host variables

	h_a = (float*)malloc(bytes);
	h_b = (float*)malloc(bytes);
	h_c = (float*)malloc(bytes);


	hipMalloc((void **)&d_a, bytes);
	hipMalloc((void **)&d_b, bytes);
	hipMalloc((void **)&d_c, bytes);

	// accepting random elements for vectors h_a and h_b

	time_t t;
	srand((unsigned)time(&t));
	int x, y, flag;

	for (unsigned i = 0 ; i < n ; i++)
	{
		x = rand()%n;

		flag=0;
		for(int j=0;j<i;j++)
		{
			if(h_a[j]==x)
			{
				i--;
				flag=1;
				break;
			}
		}
		if(flag==0)
			h_a[i]=x;
	}

	for (unsigned i = 0 ; i < n ; i++)
	{
		y = rand()%n;

		flag=0;
		for(int j=0;j<i;j++)
		{
			if(h_b[j]==y)
			{
				i--;
				flag=1;
				break;
			}
		}
		if(flag==0)
			h_b[i]=y;
	}

	/*
	printf("\nThe vector A is: \n"); 
	for(int i = 0; i < n; i++)
	{
		printf("%f\n", h_a[i]);
		
	}

	printf("\n\nThe vector B is: \n"); 
	for(int i = 0; i < n; i++)
	{
		printf("%f\n", h_b[i]);
		
	}  
	*/

	// copying the host variables onto the device for addition	
	
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	int number_of_threads_per_block = 256;

	int number_blocks = (int)ceil((float)n / number_of_threads_per_block);

	addition<<<number_blocks, number_of_threads_per_block>>>(d_a, d_b, d_c, n);

	// copying the final answer from the device to the host
	
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	printf("\n\nThe vector C after addition of A and B is: \n");
	for(int i = 0; i < n; i++)
	{
		printf("%f\n", h_c[i]);
	} 

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(h_a);
	free(h_b);
	free(h_c);

	return 0;

}
